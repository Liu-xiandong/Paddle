#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <math.h>
#include <limits>
#include <string>
#include <vector>
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/framework/op_registry.h"
#if defined(PADDLE_WITH_CUDA)
#include "paddle/fluid/platform/dynload/hipsparse.h"
#endif

namespace ops = paddle::operators;
namespace plf = paddle::platform;

namespace paddle {
namespace operators {

#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 11020
template <typename T>
__forceinline__ __device__ T CudaShuffleXorSync(unsigned mask, T val,
                                                int width = warpSize) {
  return __shfl_xor_sync(mask, val, width);
}

template <typename T, int BatchSize, int WarpSize>
__device__ __forceinline__ void WarpReduceSum(T* sum) {
#pragma unroll
  for (int offset = WarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < BatchSize; ++i) {
      T sum_val = CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = sum[i] + sum_val;
    }
  }
}

template <typename T, int BatchSize, int WarpSize>
__device__ __forceinline__ void WarpReduceMax(T* sum) {
#pragma unroll
  for (int offset = WarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < BatchSize; ++i) {
      T max_val = CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = max(sum[i], max_val);
    }
  }
}

template <typename T, int BlockSize, int NnzBlockMax, bool KpMode = false,
          bool AttnMode = false>
__global__ void BlockSparseSoftmaxForward(T* softmax, const T* src, T scale,
                                          const T* kp_mask, const T* attn_mask,
                                          const int* layout_rowptr,
                                          const int* layout_colindex,
                                          int seq_len) {
  // current thread related info
  const int WarpSize = 32;
  const int cur = blockIdx.x * blockDim.y + threadIdx.y;
  if (cur < seq_len) {
    const int cur_seqb = cur / BlockSize;
    const int cur_nnzb = layout_rowptr[cur_seqb + 1] - layout_rowptr[cur_seqb];

    T srcdata[(BlockSize * NnzBlockMax + WarpSize - 1) / WarpSize];
    T attndata[(BlockSize * NnzBlockMax + WarpSize - 1) / WarpSize];

    // read kp mask
    T datakp_mask = (KpMode == true) ? kp_mask[cur] : 0;

    // read tensor data, attn mask
    const int Iter = (cur_nnzb + WarpSize - 1) / WarpSize;
    const T* srcptr = src + layout_rowptr[cur_seqb];
    const T* attnptr = attn_mask + cur_seqb * seq_len;
    const int* colindex = layout_colindex + layout_rowptr[cur_seqb];
    for (int j = 0; j < Iter; j++) {
      int xidx = j * WarpSize + threadIdx.x;
      int didx = j;
      if (xidx < cur_nnzb) {
        if (AttnMode == true) {
          if (std::abs(attnptr[colindex[xidx]]) <
              std::numeric_limits<T>::epsilon()) {
            srcdata[didx] =
                -std::numeric_limits<T>::infinity() * scale + datakp_mask;
          } else {
            srcdata[didx] = scale * srcptr[xidx] + datakp_mask;
          }
        } else {
          srcdata[didx] = scale * srcptr[xidx] + datakp_mask;
        }
      } else {
        srcdata[didx] = -std::numeric_limits<T>::infinity();
      }
    }

    // max value
    T max_value = srcdata[0];
    const int kIteration = (cur_nnzb * BlockSize + WarpSize - 1) / WarpSize;
#pragma unroll
    for (int it = 1; it < kIteration; ++it) {
      max_value = (max_value > srcdata[it]) ? max_value : srcdata[it];
    }
    WarpReduceMax<T, 1, WarpSize>(&max_value);

    // exp sum
    T sum = 0;
#pragma unroll
    for (int it = 0; it < kIteration; ++it) {
      srcdata[it] = std::exp(srcdata[it] - max_value);
      sum += srcdata[it];
    }
    WarpReduceSum<T, 1, WarpSize>(&sum);

    // compute softmax and write out
    T* softmaxptr = softmax + layout_rowptr[cur_seqb];
    for (int j = 0; j < Iter; j++) {
      int xidx = j * WarpSize + threadIdx.x;
      int didx = j;
      if (xidx < cur_nnzb) {
        softmaxptr[xidx] = srcdata[didx] / sum;
      }
    }
  }
}

template <typename T, int BlockSize, int NnzBlockMax>
__global__ void BlockSparseSoftmaxBackward(T* dst, const T* grad, const T* src,
                                           T scale, const int* layout_rowptr,
                                           const int* layout_colindex,
                                           int seq_len) {
  // current thread related info
  const int WarpSize = 32;
  const int cur = blockIdx.x * blockDim.y + threadIdx.y;
  if (cur < seq_len) {
    const int cur_seqb = cur / BlockSize;
    const int cur_nnzb = layout_rowptr[cur_seqb + 1] - layout_rowptr[cur_seqb];

    T srcdata[(BlockSize * NnzBlockMax + WarpSize - 1) / WarpSize];
    T graddata[(BlockSize * NnzBlockMax + WarpSize - 1) / WarpSize];

    // read tensor data, attn mask
    const int Iter = (cur_nnzb + WarpSize - 1) / WarpSize;
    const T* srcptr = src + layout_rowptr[cur_seqb];
    const T* gradptr = grad + layout_rowptr[cur_seqb];
    for (int j = 0; j < Iter; j++) {
      int xidx = j * WarpSize + threadIdx.x;
      int didx = j;
      if (xidx < cur_nnzb) {
        srcdata[didx] = srcptr[xidx];
        graddata[didx] = gradptr[xidx];
      } else {
        srcdata[didx] = 0;
        graddata[didx] = 0;
      }
    }

    T sum = 0;
    const int kIteration = (cur_nnzb * BlockSize + WarpSize - 1) / WarpSize;
#pragma unroll
    for (int it = 0; it < kIteration; ++it) {
      sum += srcdata[it] * graddata[it];
    }
    WarpReduceSum<T, 1, WarpSize>(&sum);

    // compute softmax and write out
    T* dstptr = dst + layout_rowptr[cur_seqb];
    for (int j = 0; j < Iter; j++) {
      int xidx = j * WarpSize + threadIdx.x;
      int didx = j;
      if (xidx < cur_nnzb) {
        dstptr[xidx] = scale * srcdata[didx] * (graddata[didx] - sum);
      }
    }
  }
}

using Tensor = framework::Tensor;
/*
input: sparse C in CSR format (num_rows,num_rows)
output: sparse C after softmax operation
*/
template <typename DeviceContext, typename T>
void SparseSoftmaxForward(const platform::CUDADeviceContext& ctx,
                          const Tensor* offset, const Tensor* columns,
                          Tensor* input, Tensor* output, const int blocksize,
                          const int num_rows, const int num_cols) {
  const int* offset_data = offset->data<int>();
  const int* columns_data = columns->data<int>();
  T* input_data = input->data<T>();
  T* output_data = output->data<T>();

  const int BlockSize = 1;
  dim3 blocks(32, 4, 1);
  int grid = (num_rows * BlockSize + 3) / 4;
  T scaling = static_cast<T>(1.0) / sqrt(static_cast<double>(num_cols));

  const int NnzBlockMax = 256;
  BlockSparseSoftmaxForward<T, BlockSize, NnzBlockMax><<<grid, blocks>>>(
      output_data, input_data, scaling, NULL, NULL, offset_data, columns_data,
      num_rows);
}

template <typename DeviceContext, typename T>
void SparseSoftmaxBackward(const platform::CUDADeviceContext& ctx,
                           const Tensor* offset, const Tensor* columns,
                           Tensor* dx, const Tensor* dout, const Tensor* out,
                           const int blocksize, const int num_rows,
                           const int num_cols) {
  const int* offset_data = offset->data<int>();
  const int* columns_data = columns->data<int>();
  T* dx_data = dx->data<T>();
  const T* dout_data = dout->data<T>();
  const T* out_data = out->data<T>();

  const int BlockSize = 1;
  dim3 blocks(32, 4, 1);
  int grid = (num_rows * BlockSize + 3) / 4;
  T scaling = static_cast<T>(1.0) / sqrt(static_cast<double>(num_cols));

  const int NnzBlockMax = 256;
  BlockSparseSoftmaxBackward<T, BlockSize, NnzBlockMax><<<grid, blocks>>>(
      dx_data, dout_data, out_data, scaling, offset_data, columns_data,
      num_rows);
}

int GetGpuType(std::string data_type) {
  int gpu_type = -1;
  if (data_type == "float") {
    gpu_type = 0;
  } else if (data_type == "double") {
    gpu_type = 1;
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "Not support tensor type in sparse_attention OP: %s", data_type));
  }
  return gpu_type;
}

/*
input: dense A (num_rows,num_cols), dense B (num_rows,num_cols)
output: sparse C in CSR format (num_rows,num_rows)
*/
template <typename DeviceContext, typename T>
void DotSdd(const platform::CUDADeviceContext& ctx, const Tensor* A,
            const Tensor* B, const Tensor* C_offset, const Tensor* C_columns,
            Tensor* C_value, const int num_rows, const int num_cols,
            const bool A_transpose, const bool B_transpose) {
  const T* A_data = A->data<T>();
  const T* B_data = B->data<T>();
  const int* C_offset_data = C_offset->data<int>();
  const int* C_columns_data = C_columns->data<int>();
  T* C_value_data = C_value->data<T>();

  std::string data_type = framework::DataTypeToString(C_value->type());
  int gpu_type = GetGpuType(data_type);

  hipsparseHandle_t handle = NULL;
  hipsparseDnMatDescr_t matA, matB;
  hipsparseSpMatDescr_t matC;
  platform::dynload::hipsparseCreate(&handle);

  // Create dense matrix A
  platform::dynload::hipsparseCreateDnMat(
      &matA, num_rows, num_cols, num_cols, const_cast<T*>(A_data),
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F, HIPSPARSE_ORDER_ROW);
  // Create dense matrix B
  platform::dynload::hipsparseCreateDnMat(
      &matB, num_rows, num_cols, num_cols, const_cast<T*>(B_data),
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F, HIPSPARSE_ORDER_ROW);
  // Create sparse matrix C in CSR format
  int C_nnz = C_columns->dims()[1];
  platform::dynload::hipsparseCreateCsr(
      &matC, num_rows, num_rows, C_nnz, const_cast<int*>(C_offset_data),
      const_cast<int*>(C_columns_data), C_value_data, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F);

  T alpha = 1;
  T beta = 0;

  void* dBuffer = NULL;
  size_t bufferSize = 0;
  platform::dynload::hipsparseSDDMM_bufferSize(
      handle, A_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                          : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      B_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                  : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, matA, matB, &beta, matC, gpu_type == 0 ? HIP_R_32F : HIP_R_64F,
      HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);

  platform::dynload::hipsparseSDDMM(
      handle, A_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                          : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      B_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                  : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, matA, matB, &beta, matC, gpu_type == 0 ? HIP_R_32F : HIP_R_64F,
      HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);

  platform::dynload::hipsparseDestroyDnMat(matA);
  platform::dynload::hipsparseDestroyDnMat(matB);
  platform::dynload::hipsparseDestroySpMat(matC);
  platform::dynload::hipsparseDestroy(handle);
  hipFree(dBuffer);
}

/*
input: sparse A in CSR format (num_rows,num_rows), dense B (num_rows,num_cols)
output: dense C (num_rows,num_cols)
*/
template <typename DeviceContext, typename T>
void DotDsd(const platform::CUDADeviceContext& ctx, const Tensor* A_offset,
            const Tensor* A_columns, const Tensor* A_value, const Tensor* B,
            Tensor* C, const int num_rows, const int num_cols,
            const bool A_transpose, const bool B_transpose) {
  const int* A_offset_data = A_offset->data<int>();
  const int* A_columns_data = A_columns->data<int>();
  const T* A_value_data = A_value->data<T>();
  const T* B_data = B->data<T>();
  T* C_data = C->data<T>();

  std::string data_type = framework::DataTypeToString(C->type());
  int gpu_type = GetGpuType(data_type);

  hipsparseHandle_t handle = NULL;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  platform::dynload::hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  int A_nnz = A_columns->dims()[1];
  platform::dynload::hipsparseCreateCsr(
      &matA, num_rows, num_rows, A_nnz, const_cast<int*>(A_offset_data),
      const_cast<int*>(A_columns_data), const_cast<T*>(A_value_data),
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F);

  // Create dense matrix B
  platform::dynload::hipsparseCreateDnMat(
      &matB, num_rows, num_cols, num_cols, const_cast<T*>(B_data),
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F, HIPSPARSE_ORDER_ROW);
  // Create dense matrix C
  platform::dynload::hipsparseCreateDnMat(
      &matC, num_rows, num_cols, num_cols, C_data,
      gpu_type == 0 ? HIP_R_32F : HIP_R_64F, HIPSPARSE_ORDER_ROW);

  T alpha = 1;
  T beta = 0;

  void* dBuffer = NULL;
  size_t bufferSize = 0;
  // allocate an external buffer if needed
  platform::dynload::hipsparseSpMM_bufferSize(
      handle, A_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                          : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      B_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                  : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, matA, matB, &beta, matC, gpu_type == 0 ? HIP_R_32F : HIP_R_64F,
      HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);

  platform::dynload::hipsparseSpMM(
      handle, A_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                          : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      B_transpose ? HIPSPARSE_OPERATION_TRANSPOSE
                  : HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, matA, matB, &beta, matC, gpu_type == 0 ? HIP_R_32F : HIP_R_64F,
      HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);

  platform::dynload::hipsparseDestroySpMat(matA);
  platform::dynload::hipsparseDestroyDnMat(matB);
  platform::dynload::hipsparseDestroyDnMat(matC);
  platform::dynload::hipsparseDestroy(handle);
  hipFree(dBuffer);
}

std::vector<Tensor> GetSplitTensor(Tensor* input) {
  auto dims = input->dims();
  int batch_size = dims[0];
  int num_heads = dims[1];
  std::vector<int> NewDims(dims.size() - 1);
  NewDims[0] = batch_size * num_heads;
  for (int i = 1; i < NewDims.size(); i++) {
    NewDims[i] = dims[i + 1];
  }
  input->Resize(framework::make_ddim(NewDims));
  return input->Split(1, 0);
}
#endif

template <typename DeviceContext, typename T>
class SparseAttentionCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 11020
    auto query = *ctx.Input<Tensor>("Q");
    auto key = *ctx.Input<Tensor>("K");
    auto value = *ctx.Input<Tensor>("V");
    auto offset = *ctx.Input<Tensor>("offset");
    auto columns = *ctx.Input<Tensor>("columns");
    auto outputPtr = ctx.Output<Tensor>("Out");
    outputPtr->mutable_data<T>(ctx.GetPlace());
    auto ResultSddPtr = ctx.Output<Tensor>("ResultSdd");
    ResultSddPtr->mutable_data<T>(ctx.GetPlace());
    auto ResultSoftmaxPtr = ctx.Output<Tensor>("ResultSoftmax");
    ResultSoftmaxPtr->mutable_data<T>(ctx.GetPlace());

    auto output = *outputPtr;
    auto result_sdd = *ResultSddPtr;
    auto result_softmax = *ResultSoftmaxPtr;

    auto query_dims = query.dims();
    int batch_size = query_dims[0];
    int num_heads = query_dims[1];
    int M = query_dims[2];
    int N = query_dims[3];

    std::vector<Tensor> query_lists = GetSplitTensor(&query);
    std::vector<Tensor> key_lists = GetSplitTensor(&key);
    std::vector<Tensor> value_lists = GetSplitTensor(&value);
    std::vector<Tensor> offset_lists = GetSplitTensor(&offset);
    std::vector<Tensor> columns_lists = GetSplitTensor(&columns);
    std::vector<Tensor> result_sdd_lists = GetSplitTensor(&result_sdd);
    std::vector<Tensor> result_softmax_lists = GetSplitTensor(&result_softmax);
    std::vector<Tensor> output_lists = GetSplitTensor(&output);

    const auto& dev_ctx = ctx.cuda_device_context();
    const int iter_num = batch_size * num_heads;
    for (int i = 0; i < iter_num; i++) {
      DotSdd<DeviceContext, T>(dev_ctx, &query_lists[i], &key_lists[i],
                               &offset_lists[i], &columns_lists[i],
                               &result_sdd_lists[i], M, N, false, true);

      SparseSoftmaxForward<DeviceContext, T>(
          dev_ctx, &offset_lists[i], &columns_lists[i], &result_sdd_lists[i],
          &result_softmax_lists[i], 1, M, N);

      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &result_softmax_lists[i], &value_lists[i],
                               &output_lists[i], M, N, false, false);
    }
#else
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The sparse_attention OP needs to use Nvidia GPU, and the CUDA version "
        "cannot be less than 11.2"));
#endif
  }
};

template <typename DeviceContext, typename T>
class SparseAttentionGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 11020
    auto query = *ctx.Input<Tensor>("Q");
    auto key = *ctx.Input<Tensor>("K");
    auto value = *ctx.Input<Tensor>("V");
    auto offset = *ctx.Input<Tensor>("offset");
    auto columns = *ctx.Input<Tensor>("columns");
    auto ResultSdd = *ctx.Input<Tensor>("ResultSdd");
    auto ResultSoftmax = *ctx.Input<Tensor>("ResultSoftmax");
    auto dout = *ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dQueryPtr = ctx.Output<Tensor>(framework::GradVarName("Q"));
    auto* dKeyPtr = ctx.Output<Tensor>(framework::GradVarName("K"));
    auto* dValuePtr = ctx.Output<Tensor>(framework::GradVarName("V"));
    dQueryPtr->mutable_data<T>(ctx.GetPlace());
    dKeyPtr->mutable_data<T>(ctx.GetPlace());
    dValuePtr->mutable_data<T>(ctx.GetPlace());
    auto dQuery = *dQueryPtr;
    auto dKey = *dKeyPtr;
    auto dValue = *dValuePtr;

    auto query_dims = query.dims();
    int batch_size = query_dims[0];
    int num_heads = query_dims[1];
    int M = query_dims[2];
    int N = query_dims[3];

    std::vector<Tensor> query_lists = GetSplitTensor(&query);
    std::vector<Tensor> key_lists = GetSplitTensor(&key);
    std::vector<Tensor> value_lists = GetSplitTensor(&value);
    std::vector<Tensor> offset_lists = GetSplitTensor(&offset);
    std::vector<Tensor> columns_lists = GetSplitTensor(&columns);
    std::vector<Tensor> ResultSdd_lists = GetSplitTensor(&ResultSdd);
    std::vector<Tensor> ResultSoftmax_lists = GetSplitTensor(&ResultSoftmax);
    std::vector<Tensor> dout_lists = GetSplitTensor(&dout);
    std::vector<Tensor> dQuery_lists = GetSplitTensor(&dQuery);
    std::vector<Tensor> dKey_lists = GetSplitTensor(&dKey);
    std::vector<Tensor> dValue_lists = GetSplitTensor(&dValue);

    const int iter_num = batch_size * num_heads;
    const auto& dev_ctx = ctx.cuda_device_context();
    for (int i = 0; i < iter_num; i++) {
      // dValue = transpose(result_softmax) * dOut
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &ResultSoftmax_lists[i], &dout_lists[i],
                               &dValue_lists[i], M, N, true, false);

      // dResultSoftmax = dOut * transpose(Value)
      int nnz_num = columns.dims()[0];
      Tensor dResultSoftmax;
      dResultSoftmax.Resize({nnz_num});
      dResultSoftmax.mutable_data<T>(ctx.GetPlace());
      DotSdd<DeviceContext, T>(dev_ctx, &dout_lists[i], &value_lists[i],
                               &offset_lists[i], &columns_lists[i],
                               &dResultSoftmax, M, N, false, true);

      // dResultSdd = dResultSoftmax * softmax'(ResultSdd)
      Tensor dResultSdd;
      dResultSdd.Resize({nnz_num});
      dResultSdd.mutable_data<T>(ctx.GetPlace());
      SparseSoftmaxBackward<DeviceContext, T>(
          dev_ctx, &offset_lists[i], &columns_lists[i], &dResultSdd,
          &dResultSoftmax, &ResultSoftmax_lists[i], 1, M, N);

      // dQuery = dResultSdd * Key
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &dResultSdd, &key_lists[i], &dQuery_lists[i], M,
                               N, false, false);

      // dKey = transpose(dResultSdd) * Query
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &dResultSdd, &query_lists[i], &dKey_lists[i], M,
                               N, true, false);
    }
#else
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The sparse_attention OP needs to use Nvidia GPU, and the CUDA version "
        "cannot be less than 11.2"));
#endif
  }
};

}  // namespace operators
}  // namespace paddle
REGISTER_OP_CUDA_KERNEL(
    sparse_attention,
    ops::SparseAttentionCUDAKernel<plf::CUDADeviceContext, float>,
    ops::SparseAttentionCUDAKernel<plf::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    sparse_attention_grad,
    ops::SparseAttentionGradCUDAKernel<plf::CUDADeviceContext, float>,
    ops::SparseAttentionGradCUDAKernel<plf::CUDADeviceContext, double>);
